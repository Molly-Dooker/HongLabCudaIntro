﻿#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>

#include "hip/hip_runtime.h"


using namespace std;

template<typename T>
void printVector(const vector<T>& a)
{
	for (int v : a)
		cout << setw(3) << v;
	cout << endl;
}

__global__ void addKernel(const int* a, const int* b, int* c, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x; 
	if (i < size)
		c[i] = a[i] + b[i];
}
int main()
{
	const int size = (1<<22)+3;
	// 생각해볼 점: 블럭이 몇 개가 필요할까?

	vector<int> a(size);
	vector<int> b(size);
	vector<int> c_single(size);     // 결과 확인용
	vector<int> c(size, -1); // CUDA에서 계산한 결과 저장

	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
		c_single[i] = a[i] + b[i];
	}

	cout << "Add vectors using CUDA" << endl;

	{
		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		hipMalloc((void**)&dev_a, size * sizeof(int)); // input a
		hipMalloc((void**)&dev_b, size * sizeof(int)); // input b
		hipMalloc((void**)&dev_c, size * sizeof(int)); // output c

		hipEvent_t start, stop;// 시간 측정을 위한 CUDA 이벤트 생성 (시간측정도 Nsight로 할 수 있습니다.)
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0); // 시작 시간 기록

		hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice);

		const unsigned TPB = 512; // 256이나 512 권장(하드웨어, 점유율에 따라)
		const unsigned blocks = (size + TPB - 1) / TPB; // 올림
		cout << "size: "<<size<<" blocks: "<<blocks<<endl;
		addKernel<<<blocks, TPB>>>(dev_a, dev_b, dev_c, size);

		hipMemcpy(c.data(), dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

		hipEventRecord(stop, 0);  // 끝나는 시간 기록

		hipDeviceSynchronize();       // kernel이 끝날때까지 대기 (동기화)
		// cudaEventSynchronize(stop); // 불필요 (동기화 중복)

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop); // 걸린 시간 계산
		cout << "Time elapsed: " << milliseconds << " ms" << endl;

		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		hipDeviceReset();
	}

	if (size < 40) { // size가 작을 경우에는 출력해서 확인
		printVector(a);
		printVector(b);
		printVector(c_single);
		printVector(c);
	}

	for (int i = 0; i < size; i++)
		if (c_single[i] != c[i])
		{
			cout << "Wrong result" << endl;
			return 1;
		}

	cout << "Correct" << endl;

	return 0;
}

