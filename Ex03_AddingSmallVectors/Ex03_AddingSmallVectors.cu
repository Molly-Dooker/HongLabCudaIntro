﻿#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>

#include "hip/hip_runtime.h"


using namespace std;

template<typename T>
void printVector(const vector<T>& a)
{
	for (int v : a)
		cout << setw(3) << v;
	cout << endl;
}

__global__ void addKernel(const int* a, const int* b, int* c)
{
	int i = threadIdx.x;
    c[i] = a[i]+b[i];
	printf("ThreadIdx(% u, % u, % u)\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	int size = 10; // 블럭(block) 하나만으로 계산할 수 있는 크기 = deviceProp.maxThreadsPerBlock = 1024

	vector<int> a(size);
	vector<int> b(size);
	vector<int> cSingle(size); // 결과 확인용
	vector<int> c(size, -1);    // CUDA에서 계산한 결과 저장

	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
		cSingle[i] = a[i] + b[i];
	}

	cout << "Add vectors using CUDA" << endl;

	{
		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		hipMalloc((void**)&dev_a, size * sizeof(int)); // input a
		hipMalloc((void**)&dev_b, size * sizeof(int)); // input b
		hipMalloc((void**)&dev_c, size * sizeof(int)); // output c

		hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice);

		// 블럭 1개 * 쓰레드 size개
		addKernel <<<1, size >>> (dev_a, dev_b, dev_c);
		// addKernel<<<블럭이 몇 개 인지, 각 블럭당 쓰레드가 몇 개인지 >>>(dev_a, dev_b, dev_c);

		// 안내:
		// - cudaMemcpy()와 달리 커널 호출은 항상 비동기적(asynchronous)입니다. 
		// - GPU에게 명령만 내리고 CPU는 바로 다음 명령을 수행한다는 의미입니다.
		// - CPU에게 GPU가 일을 다 끝날때까지 강제로 기다리게 하고 싶다면 아래의 
		// - cudaDeviceSynchronize()를 사용할 수 있습니다.
		// - 함수 이름에서 볼 수 있듯이, 이렇게 기다리는 것을 "동기화(synchronize)"라고 합니다.

		hipDeviceSynchronize();       // kernel이 끝날때까지 대기 (동기화)

		// 안내: kernel 실행 후 cudaGetLastError() 생략하였습니다.

		// 결과 복사 device -> host
		hipMemcpy(c.data(), dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		hipDeviceReset();
	}

	if (size < 40) { // size가 작을 경우에는 출력해서 확인
		printVector(a);
		printVector(b);
		printVector(cSingle);
		printVector(c);
	}

	for (int i = 0; i < size; i++)
		if (cSingle[i] != c[i])
		{
			cout << "Wrong result" << endl;
			return 1;
		}

	cout << "Correct" << endl;
	fflush(stdout);
	return 0;
}
